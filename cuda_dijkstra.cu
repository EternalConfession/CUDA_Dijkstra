#include "hip/hip_runtime.h"
/* Name: ZHANG Jianhui
 * ID: 20374500
 * Email:
 */

/*
 * This is code skeleton for COMP5112-17Spring assignment4
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_dijkstra cuda_dijkstra_skeleton.cu
 * Run: ./cuda_dijkstra -n <number of threads> -i <input file>,
 * you will find the output in 'output.txt' file
 *
 *  by Lipeng WANG, 5th Apr 2017
 */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <vector>
#include <climits>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <time.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;
using std::vector;
using std::ceil;
using std::memcpy;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
 {                                                                              \
     const hipError_t error = call;                                            \
     if (error != hipSuccess)                                                  \
     {                                                                          \
         fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
         fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                 hipGetErrorString(error));                                    \
         exit(1);                                                               \
     }                                                                          \
 }

/*
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and one matrix dimension convert(2D->1D) function
 */
namespace utils {
    int num_threads; //number of thread
    int N; //number of vertices
    int *mat; // the adjacency matrix

    string filename; // input file name
    string outputfile; //output file name, default: 'output.txt'

    void print_usage() {
        cout << "Usage:\n" << "\tcuda_dijkstra -n <number of threads per block> -i <input file>" << endl;
        exit(0);
    }

    int parse_args(int argc, char **argv) {
        filename = "";
        outputfile = "output.txt";
        num_threads = 0;

        int opt;
        if (argc < 2) {
            print_usage();
        }
        while ((opt = getopt(argc, argv, "n:i:o:h")) != EOF) {
            switch (opt) {
                case 'n':
                    num_threads = atoi(optarg);
                    break;
                case 'i':
                    filename = optarg;
                    break;
                case 'o':
                    outputfile = optarg;
                    break;
                case 'h':
                case '?':
                default:
                    print_usage();
            }
        }
        if (filename.length() == 0 || num_threads == 0)
            print_usage();
        return 0;
    }

    /*
     * convert 2-dimension coordinate to 1-dimension
     */
    int convert_dimension_2D_1D(int x, int y) {
        return x * N + y;
    }

    int read_file(string filename) {
        std::ifstream inputf(filename, std::ifstream::in);
        inputf >> N;
        assert(N < (1024 * 1024 *
                    20)); // input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
        mat = (int *) malloc(N * N * sizeof(int));
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++) {
                inputf >> mat[convert_dimension_2D_1D(i, j)];
            }

        return 0;
    }

    string format_path(int i, int *pred) {
        string out("");
        int current_vertex = i;
        while (current_vertex != 0) {
            string s = std::to_string(current_vertex);
            std::reverse(s.begin(), s.end());
            out = out + s + ">-";
            current_vertex = pred[current_vertex];
        }
        out = out + std::to_string(0);
        std::reverse(out.begin(), out.end());
        return out;
    }

    int print_result(int *dist, int *pred) {
        std::ofstream outputf(outputfile, std::ofstream::out);
        outputf << dist[0];
        for (int i = 1; i < N; i++) {
            outputf << " " << dist[i];
        }
        for (int i = 0; i < N; i++) {
            outputf << "\n";
            if (dist[i] >= 1000000) {
                outputf << "NO PATH";
            } else {
                outputf << format_path(i, pred);
            }
        }
        outputf << endl;
        return 0;
    }
}//namespace utils


//------You may add helper functions and global variables here------

/*
 * function: find the local minimum for each block and store them to d_local_min and d_local_min_index
 * parameters: N: input size, *d_visit: array to record which vertex has been visited, *d_all_dist: array to store the distance,
 *        *d_local_min: array to store the local minimum value for each block, *d_local_min_index: array to store the local minimum index for each block
 */
__global__ void FindLocalMin(int N, int *d_visit, int *d_all_dist, int *d_local_min, int *d_local_min_index) {
    int num_per_thread = (int) ceil(N * 1.0/(blockDim.x * gridDim.x));

    __shared__ int thread_local_min[1024];
    __shared__ int thread_local_min_index[1024];

    int start = (blockIdx.x * blockDim.x + threadIdx.x) * num_per_thread;
    int end = start + num_per_thread;
    if (end > N) {
        end = N;
    }

    int thread_min = INF + 1;
    int thread_min_index = -1;
    for (int i = start; i < end; i++)
    {
        if (d_all_dist[i] < thread_min && !d_visit[i])
        {
            thread_min = d_all_dist[i];
            thread_min_index = i;
        }
    }
    thread_local_min[threadIdx.x] = thread_min;
    thread_local_min_index[threadIdx.x] = thread_min_index;
    __syncthreads();

    if (threadIdx.x == 0) {
        int block_min = INF + 1;
        int block_min_index = -1;
        for (int i = 0; i < blockDim.x; i++)
        {
            if (thread_local_min[i] < block_min)
            {
                block_min = thread_local_min[i];
                block_min_index = thread_local_min_index[i];
            }
        }
        d_local_min[blockIdx.x] = block_min;
        d_local_min_index[blockIdx.x] = block_min_index;
    }
}

/*
 * function: update the global minimum value(and index), store them to a global memory address
 * parameters: *global_min: memory address to store the global min value, *global_min_index: memory address to store the global min index
 *        *d_local_min: array stores the local min value od each block, *d_local_min_index: array stores the local min index of each block
 *        *d_visit: array stores the status(visited/un-visited) for each vertex
 */
__global__ void
UpdateGlobalMin(int *global_min, int *global_min_index, int *d_local_min, int *d_local_min_index, int *d_visit) {
    if(blockIdx.x == 0 && threadIdx.x == 0)
    {
        int global_min_tmp = INF + 1;
        int global_index_tmp = -1;
        for (int i = 0; i < gridDim.x; i++)
        {
            if(global_min_tmp > d_local_min[i])
            {
                global_min_tmp = d_local_min[i];
                global_index_tmp = d_local_min_index[i];
            }
        }
        global_min[0] = global_min_tmp;
        global_min_index[0] = global_index_tmp;
        d_visit[global_min_index[0]] = true;
    }
}

/*
 * function: update the shortest path for every un-visited vertices
 * parameters: N: input size, *mat: input matrix, *d_visit: array stores the status(visited/un-visited) for each vertex
 *             *d_all_dist: array stores the shortest distance for each vertex, *d_all_pred: array stores the predecessors
 *             *global_min: memory address that stores the global min value, *global_min_index: memory address that stores the global min index
 */
__global__ void
UpdatePath(int N, int *mat, int *d_visit, int *d_all_dist, int *d_all_pred, int *global_min, int *global_min_index) {
    int num_per_thread = (int) ceil(N * 1.0/(blockDim.x * gridDim.x));

    int start = (blockIdx.x * blockDim.x + threadIdx.x) * num_per_thread;
    int end = start + num_per_thread;
    if (end > N) {
        end = N;
    }

    for(int i = start; i < end; i++)
    {
        if (global_min[0] + mat[global_min_index[0] * N + i] < d_all_dist[i])
        {
            d_all_dist[i] = global_min[0] + mat[global_min_index[0] * N + i];
            d_all_pred[i] = global_min_index[0];
        }
    }
    __syncthreads();
}

//Do not change anything below this line
void dijkstra(int N, int p, int *mat, int *all_dist, int *all_pred) {

    //threads number for each block should smaller than or equal to 1024
    assert(p <= 1024);

    //we restrict this value to 8, DO NOT change it!
    int blocksPerGrid = 8;

    //NOTICE: (p * 8) may LESS THAN N
    int threadsPerBlock = p;

    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);


    //allocate memory
    int *h_visit;
    int *d_mat, *d_visit, *d_all_dist, *d_all_pred, *d_local_min, *d_local_min_index;
    int *d_global_min, *d_global_min_index;

    h_visit = (int *) calloc(N, sizeof(int));
    hipMalloc(&d_mat, sizeof(int) * N * N);
    hipMalloc(&d_visit, sizeof(int) * N);
    hipMalloc(&d_all_dist, sizeof(int) * N);
    hipMalloc(&d_all_pred, sizeof(int) * N);
    hipMalloc(&d_local_min, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_local_min_index, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_global_min, sizeof(int));
    hipMalloc(&d_global_min_index, sizeof(int));

    //initialization and copy data from host to device
    for (int i = 0; i < N; i++) {
        all_dist[i] = mat[i];
        all_pred[i] = 0;
        h_visit[i] = 0;
    }
    //h_visit[0] = 1;

    hipMemcpy(d_mat, mat, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_dist, all_dist, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_pred, all_pred, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_visit, h_visit, sizeof(int) * N, hipMemcpyHostToDevice);

    //dijkstra iterations
    for (int iter = 1; iter < N; iter++) {
        FindLocalMin <<< blocks, threads >>> (N, d_visit, d_all_dist, d_local_min, d_local_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
        //printf("%s%d\n", "Iter Num: ", iter);
        UpdateGlobalMin <<< blocks, threads >>>
                                    (d_global_min, d_global_min_index, d_local_min, d_local_min_index, d_visit);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdatePath << < blocks, threads >> >
                                (N, d_mat, d_visit, d_all_dist, d_all_pred, d_global_min, d_global_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
    }

    //copy results from device to host
    hipMemcpy(all_dist, d_all_dist, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(all_pred, d_all_pred, sizeof(int) * N, hipMemcpyDeviceToHost);

    //free memory
    free(h_visit);
    hipFree(d_mat);
    hipFree(d_visit);
    hipFree(d_all_dist);
    hipFree(d_all_pred);
    hipFree(d_local_min);
    hipFree(d_local_min_index);
    hipFree(d_global_min);
    hipFree(d_global_min_index);

}

int main(int argc, char **argv) {
    assert(utils::parse_args(argc, argv) == 0);
    assert(utils::read_file(utils::filename) == 0);

    //`all_dist` stores the distances and `all_pred` stores the predecessors
    int *all_dist;
    int *all_pred;
    all_dist = (int *) calloc(utils::N, sizeof(int));
    all_pred = (int *) calloc(utils::N, sizeof(int));

    //time counter
    timeval start_wall_time_t, end_wall_time_t;
    float ms_wall;

    hipDeviceReset();

    //start timer
    gettimeofday(&start_wall_time_t, nullptr);
    dijkstra(utils::N, utils::num_threads, utils::mat, all_dist, all_pred);
    CHECK(hipDeviceSynchronize());

    //end timer
    gettimeofday(&end_wall_time_t, nullptr);
    ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
               + end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

    std::cerr << "Time(ms): " << ms_wall << endl;

    utils::print_result(all_dist, all_pred);

    free(utils::mat);
    free(all_dist);
    free(all_pred);

    return 0;
}
